#include <iostream>
#include <hip/hip_runtime.h>
#include "gpu.hpp"

void printCudaVersion()
{
    std::cout << "CUDA Compiled version: " << __CUDACC_VER_MAJOR__ << "." << __CUDACC_VER_MINOR__ << std::endl;

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}
